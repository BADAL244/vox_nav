#include "hip/hip_runtime.h"
// Copyright (c) 2020 Fetullah Atas, Norwegian University of Life Sciences
//
// Licensed under the Apache License, Version 2.0 (the "License");
// you may not use this file except in compliance with the License.
// You may obtain a copy of the License at
//
//     http://www.apache.org/licenses/LICENSE-2.0
//
// Unless required by applicable law or agreed to in writing, software
// distributed under the License is distributed on an "AS IS" BASIS,
// WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
// See the License for the specific language governing permissions and
// limitations under the License.

#include <rclcpp/rclcpp.hpp>
#include <sensor_msgs/msg/point_cloud2.hpp>
#include <sensor_msgs/msg/imu.hpp>
#include <nav_msgs/msg/odometry.hpp>
#include "message_filters/synchronizer.h"
#include "message_filters/subscriber.h"
#include "message_filters/sync_policies/approximate_time.h"
#include <pcl/common/common.h>
#include <pcl_ros/transforms.hpp>
#include <pcl/kdtree/kdtree_flann.h>
#include <pcl/filters/crop_box.h>
#include "tf2_geometry_msgs/tf2_geometry_msgs.h"

#include <queue>

#include "cupoch_conversions/cupoch_conversions.hpp"
#include "cupoch/cupoch.h"
#include "vox_nav_cupoch_experimental/visibility_control.h"

namespace vox_nav_utilities
{

    class DynamicPoints : public rclcpp::Node
    {

    public:
        DynamicPoints();
        ~DynamicPoints();

        typedef message_filters::sync_policies::ApproximateTime<
            sensor_msgs::msg::PointCloud2,
            nav_msgs::msg::Odometry,
            sensor_msgs::msg::Imu>
            CloudOdomApprxTimeSyncPolicy;
        typedef message_filters::Synchronizer<CloudOdomApprxTimeSyncPolicy>
            CloudOdomApprxTimeSyncer;

        void cloudOdomCallback(
            const sensor_msgs::msg::PointCloud2::ConstSharedPtr &cloud,
            const nav_msgs::msg::Odometry::ConstSharedPtr &odom,
            const sensor_msgs::msg::Imu::ConstSharedPtr &imu);

        void shoot(
            std::vector<std::shared_ptr<cupoch::geometry::PointCloud>> &cloud_vector);

        void getRPYfromMsgQuaternion(
            const geometry_msgs::msg::Quaternion q_msg, double &roll, double &pitch,
            double &yaw);

    private:
        rclcpp::TimerBase::SharedPtr timer_;
        message_filters::Subscriber<sensor_msgs::msg::PointCloud2> cloud_subscriber_;
        message_filters::Subscriber<nav_msgs::msg::Odometry> odom_subscriber_;
        message_filters::Subscriber<sensor_msgs::msg::Imu> imu_subscriber_;
        std::shared_ptr<CloudOdomApprxTimeSyncer> cloud_odom_data_approx_time_syncher_;

        int sequence_horizon_;
        double dt_;
        double sensor_height_;

        std::vector<std::tuple<sensor_msgs::msg::PointCloud2::SharedPtr,
                               nav_msgs::msg::Odometry::SharedPtr,
                               sensor_msgs::msg::Imu::SharedPtr>>
            cloud_odom_vector_;

        rclcpp::Publisher<sensor_msgs::msg::PointCloud2>::SharedPtr pub_;

        rclcpp::Time last_recieved_msg_stamp_;
        rclcpp::Time stamp_;
    };

    DynamicPoints::DynamicPoints()
        : Node("dynamic_points_node")
    {
        cloud_subscriber_.subscribe(this, "points", rmw_qos_profile_sensor_data);
        odom_subscriber_.subscribe(this, "odom", rmw_qos_profile_sensor_data);
        imu_subscriber_.subscribe(this, "imu", rmw_qos_profile_sensor_data);

        declare_parameter("sequence_horizon", 0);
        get_parameter("sequence_horizon", sequence_horizon_);

        declare_parameter("dt", 0.0);
        get_parameter("dt", dt_);

        declare_parameter("sensor_height", 0.0);
        get_parameter("sensor_height", sensor_height_);

        cloud_odom_data_approx_time_syncher_.reset(
            new CloudOdomApprxTimeSyncer(
                CloudOdomApprxTimeSyncPolicy(100),
                cloud_subscriber_,
                odom_subscriber_,
                imu_subscriber_));

        cloud_odom_data_approx_time_syncher_->registerCallback(
            std::bind(
                &DynamicPoints::cloudOdomCallback, this,
                std::placeholders::_1,
                std::placeholders::_2,
                std::placeholders::_3));

        pub_ = this->create_publisher<sensor_msgs::msg::PointCloud2>(
            "merged", rclcpp::SystemDefaultsQoS());
    }

    DynamicPoints::~DynamicPoints()
    {
    }

    void DynamicPoints::cloudOdomCallback(
        const sensor_msgs::msg::PointCloud2::ConstSharedPtr &cloud,
        const nav_msgs::msg::Odometry::ConstSharedPtr &odom,
        const sensor_msgs::msg::Imu::ConstSharedPtr &imu)
    {
        last_recieved_msg_stamp_ = cloud->header.stamp;

        RCLCPP_INFO(get_logger(), "Recieved a msg");

        if (cloud_odom_vector_.size() == 0)
        {
            auto curr_cloud_odom_pair =
                std::make_tuple<>(std::make_shared<sensor_msgs::msg::PointCloud2>(*cloud),
                                  std::make_shared<nav_msgs::msg::Odometry>(*odom),
                                  std::make_shared<sensor_msgs::msg::Imu>(*imu));
            cloud_odom_vector_.push_back(curr_cloud_odom_pair);
            stamp_ = std::get<0>(cloud_odom_vector_.back())->header.stamp;
        }

        if (cloud_odom_vector_.size() < sequence_horizon_ &&
            (last_recieved_msg_stamp_ - stamp_).seconds() > dt_)
        {
            auto curr_cloud_odom_pair = std::make_tuple<>(std::make_shared<sensor_msgs::msg::PointCloud2>(*cloud),
                                                          std::make_shared<nav_msgs::msg::Odometry>(*odom),
                                                          std::make_shared<sensor_msgs::msg::Imu>(*imu));
            cloud_odom_vector_.push_back(curr_cloud_odom_pair);
            stamp_ = std::get<0>(cloud_odom_vector_.back())->header.stamp;
        }

        if (cloud_odom_vector_.size() == sequence_horizon_)
        {
            std::vector<std::shared_ptr<cupoch::geometry::PointCloud>> transformed_pcl_sequences;
            auto merged = std::make_shared<cupoch::geometry::PointCloud>();

            Eigen::Matrix4f odom_T = Eigen::Matrix4f::Identity();
            for (int i = 0; i < cloud_odom_vector_.size(); i++)
            {
                auto dist = Eigen::Vector3f(
                                std::get<1>(cloud_odom_vector_.back())->pose.pose.position.x -
                                    std::get<1>(cloud_odom_vector_[i])->pose.pose.position.x,
                                std::get<1>(cloud_odom_vector_.back())->pose.pose.position.y -
                                    std::get<1>(cloud_odom_vector_[i])->pose.pose.position.y,
                                std::get<1>(cloud_odom_vector_.back())->pose.pose.position.z -
                                    std::get<1>(cloud_odom_vector_[i])->pose.pose.position.z)
                                .norm();

                double yaw_latest, pitch_latest, roll_latest;
                double yaw, pitch, roll;

                getRPYfromMsgQuaternion(
                    std::get<2>(cloud_odom_vector_.back())->orientation,
                    roll_latest,
                    pitch_latest,
                    yaw_latest);
                getRPYfromMsgQuaternion(
                    std::get<2>(cloud_odom_vector_[i])->orientation,
                    roll,
                    pitch,
                    yaw);

                auto rot = cupoch::geometry::GetRotationMatrixFromXYZ(Eigen::Vector3f(roll_latest - roll,
                                                                                      pitch_latest - pitch,
                                                                                      yaw_latest - yaw));

                auto trans = Eigen::Vector3f(
                    dist * cos(yaw_latest - yaw),
                    dist * sin(yaw_latest - yaw),
                    sensor_height_);

                odom_T.block<3, 3>(0, 0) = rot;
                odom_T.block<3, 1>(0, 3) = trans;

                auto cupoch_pc = std::make_shared<cupoch::geometry::PointCloud>();
                cupoch_conversions::rosToCupoch(std::get<0>(cloud_odom_vector_[i]), cupoch_pc);

                cupoch_pc->Transform(odom_T.inverse());
                cupoch_pc->PaintUniformColor(Eigen::Vector3f(1, 0, 0));

                cupoch::geometry::AxisAlignedBoundingBox<3> bbx(Eigen::Vector3f(-20, -20, -4),
                                                                Eigen::Vector3f(20, 20, 4));
                cupoch_pc = cupoch_pc->Crop(bbx);
                cupoch_pc = cupoch_pc->VoxelDownSample(0.32);

                transformed_pcl_sequences.push_back(cupoch_pc);
            }

            shoot(transformed_pcl_sequences);

            cloud_odom_vector_.erase(cloud_odom_vector_.begin());
        }
    }

    void DynamicPoints::shoot(
        std::vector<std::shared_ptr<cupoch::geometry::PointCloud>> &cloud_vector)
    {
        auto source = cloud_vector[0];
        auto target = cloud_vector[1];

        // ICP
        Eigen::Matrix4f eye = Eigen::Matrix4f::Identity();
        auto point_to_point =
            cupoch::registration::TransformationEstimationPointToPoint();
        cupoch::registration::ICPConvergenceCriteria criteria;
        criteria.max_iteration_ = 1000;
        auto res = cupoch::registration::RegistrationICP(*source, *target, 5.0, eye,
                                                         point_to_point, criteria);
        source->Transform(res.transformation_);

        // REMOVE THE GROUND
        auto segmented_source = source->SegmentPlane(0.4, 3, 50);
        auto segmented_target = target->SegmentPlane(0.4, 3, 50);
        source = source->SelectByIndex(std::get<1>(segmented_source), true);
        target = target->SelectByIndex(std::get<1>(segmented_target), true);

        // REMOVE THE NOISE
        auto denoised_source = source->RemoveStatisticalOutliers(10, 0.1);
        auto denoised_target = target->RemoveStatisticalOutliers(10, 0.1);

        denoised_source =
            std::get<0>(denoised_source)->RemoveRadiusOutliers(2, 0.2);
        denoised_target =
            std::get<0>(denoised_target)->RemoveRadiusOutliers(2, 0.2);

        source = std::get<0>(denoised_source);
        target = std::get<0>(denoised_target);

        // START VOXEL STUFF
        double voxel_size = 0.32;
        auto voxel_source = cupoch::geometry::VoxelGrid::CreateFromPointCloud(
            *source, voxel_size);
        auto voxel_target = cupoch::geometry::VoxelGrid::CreateFromPointCloud(
            *target, voxel_size);

        // COMPUTE COLLISIONS
        auto uniq_target = cupoch::collision::ComputeIntersection(
                               *voxel_source, *voxel_target, 0.0)
                               ->GetSecondCollisionIndices();

        // EXTRACT ONLY VOXELS THAT ARE COLLISION FREE
        auto voxel_target_collision_free =
            std::make_shared<cupoch::geometry::VoxelGrid>();

        RCLCPP_INFO(get_logger(), "voxel_target  %d POINT", voxel_target->voxels_values_.size());
        RCLCPP_INFO(get_logger(), "uniq_target  %d POINT", uniq_target.size());

        if (!uniq_target.size())
        {
            RCLCPP_WARN(get_logger(), "Empty collision-free voxel vector, doing nothing");
            return;
        }

        voxel_target->SelectByIndexImpl(
            *voxel_target, *voxel_target_collision_free, uniq_target, true);

        RCLCPP_INFO(get_logger(), "voxel_target_collision_free  %d POINT", voxel_target_collision_free->voxels_values_.size());
        RCLCPP_INFO(get_logger(), "voxel_target_collision_free  %d POINT", voxel_target_collision_free->voxels_keys_.size());

        // EXTRACT POINTS OF COLLISION FREE VOXELS
        auto included_points_target =
            voxel_target_collision_free->CheckIfIncluded(target->points_);
        cupoch::utility::device_vector<size_t> included_points_target_indices;

        for (size_t i = 0; i < target->points_.size(); i++)
        {
            if (included_points_target[i])
            {
                included_points_target_indices.push_back(i);
            }
        }

        auto target_collision_free_cloud = target->SelectByIndex(included_points_target_indices);
        auto merged = std::make_shared<cupoch::geometry::PointCloud>(*target_collision_free_cloud);
        sensor_msgs::msg::PointCloud2 pcl_msg;
        cupoch_conversions::cupochToRos(merged, pcl_msg);
        pcl_msg.header = std::get<0>(cloud_odom_vector_.back())->header;
        pub_->publish(pcl_msg);
        cloud_vector.clear();
    }

    void DynamicPoints::getRPYfromMsgQuaternion(
        const geometry_msgs::msg::Quaternion q_msg, double &roll, double &pitch,
        double &yaw)
    {
        tf2::Quaternion q;
        tf2::fromMsg(q_msg, q);
        tf2::Matrix3x3 m(q);
        m.getRPY(roll, pitch, yaw);
    }

} // namespace

int main(int argc, char const *argv[])
{
    rclcpp::init(argc, argv);
    cupoch::utility::InitializeAllocator();
    auto node = std::make_shared<vox_nav_utilities::DynamicPoints>();
    rclcpp::spin(node);
    rclcpp::shutdown();
    return 0;
}
